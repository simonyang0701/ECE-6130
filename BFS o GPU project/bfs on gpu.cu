#include "hip/hip_runtime.h"
/*
*
* ECE 6130 Big Data and Cloud Computing
* Spring 2019
* Project code: Highly Optimized Parallel and Distributed Breadth First Search on Graphic Processing Units
* Name: Tianyu Yang
* GW ID:G38878678
* Referenced from https://siddharths2710.wordpress.com/2017/05/16/implementing-breadth-first-search-in-cuda/
*
*/

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <conio.h>
#include <iostream>
#include <ctime>
#include <ratio>
#include <chrono>

#define NUM_NODES 99999999//1023
#define parameter 511//511
#define time 1

using namespace std;
int n, r;
double d;
FILE *f;

typedef struct
{
	int start;     // Index of first adjacent node in Ea	
	int length;    // Number of adjacent nodes 
} Node;

// Define the structure of node

__global__ void CUDA_BFS_KERNEL(Node *Va, int *Ea, bool *Fa, bool *Xa, int *Ca, bool *done)
{

	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id > NUM_NODES)
		*done = false;


	if (Fa[id] == true && Xa[id] == false)
	{
		printf("%d ", id); //This printf gives the order of vertices in BFS	
		Fa[id] = false;
		Xa[id] = true;
		__syncthreads();
		int k = 0;
		int i;
		int start = Va[id].start;
		int end = start + Va[id].length;
		for (int i = start; i < end; i++)
		{
			int nid = Ea[i];

			if (Xa[nid] == false)
			{
				Ca[nid] = Ca[id] + 1;
				Fa[nid] = true;
				*done = false;
			}

		}

	}

}

// The BFS frontier corresponds to all the nodes being processed at the current level.

int main()
{
	Node node[NUM_NODES];
	//int edgesSize = 2 * NUM_NODES;
	int edges[NUM_NODES];

	int a[NUM_NODES];
	int tmp[NUM_NODES];
	char fileName[] = "web-Google.txt";
	f = fopen(fileName, "r");
	n = 0;
	while (1) {
		r = fscanf(f, "%lf", &d);
		if (1 == r) {
			n++;
			//printf("[%d]==%lg\n", n-1, d);
			a[n - 1] = (int)d;
		}
		else if (0 == r) {
			fscanf(f, "%*c");
		}
		else break;
	}

	//number of nodes and edges
	int n = a[1];
	int e = a[2];
	Node node[NUM_NODES];
	//int edgesSize = 2 * NUM_NODES;
	int edges[NUM_NODES];
	cout << "No. of nodes = " << n << endl;
	cout << "No. of edges = " <<e << endl;
	for (int i = 3; i < 2 * e + 3; i++) {
		if (i % 2 == 0) {
			edges[i / 2 - 2] = a[i];
		}
		if (i % 2 == 1) {
			tmp[(i - 1) / 2 - 1] = a[i];
		}
	}
	for (int i = 0; i < n; i++) {
		node[i].length = 0;
	}
	for (int i = 0; i < e; i++) {
		//cout << edges[i] << endl;
		//cout <<tmp[i] << endl;
		if (node[tmp[i]].start != 0) {
			node[tmp[i]].start = i;
		}
		node[tmp[i]].length++;
	}
	for (int i = 0; i < n; i++) {
		//cout << node[i].start << endl;
		//cout << node[i].length << endl;
	}
	fclose(f);

	// Special graph nodes
	/*for (int i = 0; i < parameter; i++) {
		node[i].start = 2*i;
		node[i].length = 2;
	}
	for (int i = parameter; i < NUM_NODES; i++) {
		node[i].start = i+1;
		node[i].length = 0;
	}
	for (int i = 0; i < NUM_NODES; i++) {
		edges[i] = i+1;
	}*/

	 //Eg. 1
	/*node[0].start = 0;
	node[0].length = 2;

	node[1].start = 2;
	node[1].length = 1;

	node[2].start = 3;
	node[2].length = 1;

	node[3].start = 4;
	node[3].length = 1;

	node[4].start = 5;
	node[4].length = 0;

	edges[0] = 1;
	edges[1] = 2;
	edges[2] = 4;
	edges[3] = 3;
	edges[4] = 4;*/

	// Eg. 2
	 /*node[0].start = 0;
	 node[0].length = 2;

	 node[1].start = 2;
	 node[1].length = 2;

	 node[2].start = 4;
	 node[2].length = 2;

	 node[3].start = 6;
	 node[3].length = 2;

	 node[4].start = 5;
	 node[4].length = 0;

	 edges[0] = 1;
	 edges[1] = 2;	
	 edges[2] = 0;
	 edges[3] = 3;
	 edges[4] = 0;
	 edges[5] = 3;
	 edges[6] = 1;
	 edges[7] = 2;*/

	bool frontier[NUM_NODES] = { false };
	bool visited[NUM_NODES] = { false };
	int cost[NUM_NODES] = { 0 };

	int source = 0;
	frontier[source] = true;

	Node* Va;
	hipMalloc((void**)&Va, sizeof(Node)*NUM_NODES);
	hipMemcpy(Va, node, sizeof(Node)*NUM_NODES, hipMemcpyHostToDevice);

	int* Ea;
	hipMalloc((void**)&Ea, sizeof(Node)*NUM_NODES);
	hipMemcpy(Ea, edges, sizeof(Node)*NUM_NODES, hipMemcpyHostToDevice);

	bool* Fa;
	hipMalloc((void**)&Fa, sizeof(bool)*NUM_NODES);
	hipMemcpy(Fa, frontier, sizeof(bool)*NUM_NODES, hipMemcpyHostToDevice);

	bool* Xa;
	hipMalloc((void**)&Xa, sizeof(bool)*NUM_NODES);
	hipMemcpy(Xa, visited, sizeof(bool)*NUM_NODES, hipMemcpyHostToDevice);

	int* Ca;
	hipMalloc((void**)&Ca, sizeof(int)*NUM_NODES);
	hipMemcpy(Ca, cost, sizeof(int)*NUM_NODES, hipMemcpyHostToDevice);

	int num_blks = 1;
	int threads = 5;

	bool done;
	bool* d_done;
	hipMalloc((void**)&d_done, sizeof(bool));
	printf("\n\n");
	int count = 0;

	printf("Threads Order: \n\n");

	using namespace std::chrono;
	auto start = high_resolution_clock::now();

	// Run n times for Bfs program
	for (int i = 0; i < time; i++) {
		do {
			count++;
			done = true;
			hipMemcpy(d_done, &done, sizeof(bool), hipMemcpyHostToDevice);
			CUDA_BFS_KERNEL << <num_blks, threads >> > (Va, Ea, Fa, Xa, Ca, d_done);
			hipMemcpy(&done, d_done, sizeof(bool), hipMemcpyDeviceToHost);

		} while (!done);
	}

	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	std::cout << "\nTime taken: " << duration.count() << " us" << std::endl;

	hipMemcpy(cost, Ca, sizeof(int)*NUM_NODES, hipMemcpyDeviceToHost);

	printf("\nNumber of threads used : %d \n", count);


	printf("\nThreads for each node: ");
	for (int i = 0; i < NUM_NODES; i++)
		printf("%d    ", cost[i]);
	printf("\n");
	_getch();
	system("pause");

}